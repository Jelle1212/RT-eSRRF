#include "pipeline.hu"
#include "spatial.hu"
#include "temporal.hu"
#include <hip/hip_runtime.h>
#include <stdio.h>

float *d_rgc_maps, *d_sr_image, *d_input_frame, *d_output_frame, *d_mean_image;
hipStream_t stream1, stream2, stream3, stream4;

extern "C" void initPipeline(int nFrames, int rows, int cols, int magnification) {
    int rowsM = rows * magnification;
    int colsM = cols * magnification;
    int total_pixels = rowsM * colsM;

    CHECK_CUDA(hipMalloc(&d_rgc_maps, nFrames * total_pixels * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_sr_image, total_pixels * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_input_frame, rows * cols * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output_frame, total_pixels * sizeof(float))); // Pre-allocated buffer
    CHECK_CUDA(hipMalloc(&d_mean_image, total_pixels * sizeof(float))); // Pre-allocated buffer
    CHECK_CUDA(hipStreamCreate(&stream1));
    CHECK_CUDA(hipStreamCreate(&stream2));
    CHECK_CUDA(hipStreamCreate(&stream3));
    CHECK_CUDA(hipStreamCreate(&stream4));
}

extern "C" void processFrame(const float* image_in, float* sr_image, int frame_index, int nFrames, int rows, int cols, int magnification, int shift, int radius, int sensitivity, bool doIntensityWeighting, int type) {
    int rowsM = rows * magnification;
    int colsM = cols * magnification;
    int total_pixels = rowsM * colsM;
    int buffer_offset = (frame_index % nFrames) * total_pixels;

    // Async Copy: Host to Device
    CHECK_CUDA(hipMemcpyAsync(d_input_frame, image_in, rows * cols * sizeof(float), hipMemcpyHostToDevice, stream1));

    // Spatial processing: directly writes to pre-allocated d_output_frame
    spatial(d_input_frame, d_output_frame, rows, cols, shift, magnification, radius, sensitivity, doIntensityWeighting);

    // Async Copy: Device to Device
    CHECK_CUDA(hipMemcpyAsync(d_rgc_maps + buffer_offset, d_output_frame, total_pixels * sizeof(float), hipMemcpyDeviceToDevice, stream3));

    // Trigger temporal processing only when buffer is full
    if (frame_index >= nFrames - 1) {
        temporal(d_rgc_maps, d_sr_image, d_mean_image, type, nFrames, rowsM, colsM);
        CHECK_CUDA(hipMemcpyAsync(sr_image, d_sr_image, total_pixels * sizeof(float), hipMemcpyDeviceToHost, stream4));
    }
}

extern "C" void deintPipeline() {
    CHECK_CUDA(hipFree(d_rgc_maps));
    CHECK_CUDA(hipFree(d_sr_image));
    CHECK_CUDA(hipFree(d_input_frame));
    CHECK_CUDA(hipFree(d_output_frame));
    CHECK_CUDA(hipFree(d_mean_image));

    CHECK_CUDA(hipStreamDestroy(stream1));
    CHECK_CUDA(hipStreamDestroy(stream2));
    CHECK_CUDA(hipStreamDestroy(stream3));
    CHECK_CUDA(hipStreamDestroy(stream4));
}