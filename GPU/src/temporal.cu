#include "hip/hip_runtime.h"
#include "temporal.hu"
#include <stdio.h>

#define THREADS_PER_BLOCK 256  // Adjust based on architecture

__global__ void incremental_average_kernel(const float* d_rgc_map, float* d_mean_image, int frame_idx, int frames, int total_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_pixels) {
        d_mean_image[idx] = d_mean_image[idx] + (d_rgc_map[idx] - d_mean_image[idx]) / (float)frames;
    }
}

__global__ void incremental_variance_kernel(const float* d_rgc_map, float* d_mean_image, float* d_var_image, int frame_idx, int total_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_pixels) {
        float old_mean = d_mean_image[idx];
        float alpha = 1.0f / (frame_idx + 1);
        
        // Update mean
        float new_mean = (1 - alpha) * old_mean + alpha * d_rgc_map[idx];
        d_mean_image[idx] = new_mean;

        // Update variance using Welford’s method
        d_var_image[idx] += (d_rgc_map[idx] - old_mean) * (d_rgc_map[idx] - new_mean);
    }
}

__global__ void temporal_auto_correlation_kernel(
    const float* image_stack, float* image_out, const float* mean_image, int frames, int total_pixels, int nlag) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_pixels) {
        float sum_corr = 0.0f;
        for (int f = 0; f < frames - nlag; f++) {
            float centered1 = image_stack[f * total_pixels + idx] - mean_image[idx];
            float centered2 = image_stack[(f + nlag) * total_pixels + idx] - mean_image[idx];
            sum_corr += centered1 * centered2;
        }
        image_out[idx] = sum_corr / (frames - nlag);
    }
}

extern "C" {
    void temporal(TemporalParams &params, float *d_rgc_map) {
        int total_pixels = params.rowsM * params.colsM;
        int nlag = 1;

        // Define block and grid size
        dim3 blockSize(THREADS_PER_BLOCK);
        dim3 gridSize((total_pixels + blockSize.x - 1) / blockSize.x);

        if (params.type == 0) {
            printf("%d\n",params.frame_idx);
            incremental_average_kernel<<<gridSize, blockSize>>>(d_rgc_map, params.d_sr_image, params.frame_idx, params.frames, total_pixels);
        } else if (params.type == 1) {
            incremental_variance_kernel<<<gridSize, blockSize>>>(d_rgc_map, params.d_sr_image, params.d_mean_image, params.frame_idx, total_pixels);
        } else if (params.type == 2){
            // average_kernel<<<gridSize, blockSize>>>(params.d_rgc_maps, params.d_mean_image, params.frames, total_pixels);
            // hipDeviceSynchronize();
            // temporal_auto_correlation_kernel<<<gridSize, blockSize>>>(
            //     params.d_rgc_maps, params.d_sr_image, params.d_mean_image, params.frames, total_pixels, nlag);
        } else {
            printf("ERROR: Unsupported Temporal Type: %d\n", params.type);
            return;
        }
    }
}
