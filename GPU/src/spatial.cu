#include "spatial.hu"
#include "shift_magnify.hu"
#include "roberts_cross_gradients.hu"
#include "radial_gradient_convergence.hu"
#include "settings.hu"
#include <stdio.h>
#include <stdlib.h>


extern "C" {
    void spatial(const float *d_image_in, float* d_rgc_map, int rows, int cols, 
                float shift, float magnification, float radius, 
                float sensitivity, bool doIntensityWeighting) {

        float *d_magnified_image;
        float *d_gradient_col;
        float *d_gradient_row;
        float *d_gradient_col_interp;
        float *d_gradient_row_interp;
        
        int rowsM = (int)(rows * magnification);
        int colsM = (int)(cols * magnification);

        // Allocate memory on the GPU
        hipMalloc((void**)&d_magnified_image, MAX_ROWS * MAX_COLS * sizeof(float));
        hipMalloc((void**)&d_gradient_col, MAX_INPUT_ROWS * MAX_INPUT_COLS * sizeof(float));
        hipMalloc((void**)&d_gradient_row, MAX_INPUT_ROWS * MAX_INPUT_COLS * sizeof(float));
        hipMalloc((void**)&d_gradient_col_interp, 2 * MAX_ROWS * 2 * MAX_COLS * sizeof(float));
        hipMalloc((void**)&d_gradient_row_interp, 2 * MAX_ROWS * 2 * MAX_COLS * sizeof(float));

        // Create CUDA streams
        hipStream_t stream1, stream2, stream3, stream4;
        hipStreamCreate(&stream1);
        hipStreamCreate(&stream2);
        hipStreamCreate(&stream3);
        hipStreamCreate(&stream4);

        // Call the shift_magnify function to apply shift and magnification
        shift_magnify(d_image_in, d_magnified_image, rows, cols, shift, shift, magnification, magnification, stream1);
        roberts_cross_gradients(d_image_in, d_gradient_col, d_gradient_row, rows, cols, stream2);

        // Synchronize streams to ensure both operations are complete
        hipStreamSynchronize(stream1);
        hipStreamSynchronize(stream2);

        shift_magnify(d_gradient_col, d_gradient_col_interp, rows, cols, shift, shift, magnification * 2, magnification * 2, stream3);
        shift_magnify(d_gradient_row, d_gradient_row_interp, rows, cols, shift, shift, magnification * 2, magnification * 2, stream4);
        
        // Synchronize streams to ensure both operations are complete
        hipStreamSynchronize(stream3);
        hipStreamSynchronize(stream4);
        
        radial_gradient_convergence(d_gradient_col_interp, d_gradient_row_interp, d_magnified_image, rowsM, colsM, magnification, radius, sensitivity, doIntensityWeighting, d_rgc_map);
        
        hipFree(d_magnified_image);
        hipFree(d_gradient_col);
        hipFree(d_gradient_row);
        hipFree(d_gradient_col_interp);
        hipFree(d_gradient_row_interp);

        // Destroy streams
        hipStreamDestroy(stream1);
        hipStreamDestroy(stream2);
        hipStreamDestroy(stream3);
        hipStreamDestroy(stream4);
    }
}

