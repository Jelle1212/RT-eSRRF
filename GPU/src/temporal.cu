#include "hip/hip_runtime.h"
#include "temporal.hu"
#include "settings.hu"
#include <stdio.h>

#define THREADS_PER_BLOCK 256  // Adjust based on architecture

__global__ void average_kernel(const float* image_stack, float* image_out, int frames, int total_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_pixels) {
        float sum = 0.0f;
        for (int f = 0; f < frames; f++) {
            sum += image_stack[f * total_pixels + idx];
        }
        image_out[idx] = sum / frames;
    }
}

__global__ void variance_kernel(const float* image_stack, float* image_out, const float* mean_image, int frames, int total_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_pixels) {
        float var_sum = 0.0f;
        for (int f = 0; f < frames; f++) {
            float diff = image_stack[f * total_pixels + idx] - mean_image[idx];
            var_sum += diff * diff;
        }
        image_out[idx] = var_sum / frames;
    }
}

__global__ void temporal_auto_correlation_kernel(
    const float* image_stack, float* image_out, const float* mean_image, int frames, int total_pixels, int nlag) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_pixels) {
        float sum_corr = 0.0f;
        for (int f = 0; f < frames - nlag; f++) {
            float centered1 = image_stack[f * total_pixels + idx] - mean_image[idx];
            float centered2 = image_stack[(f + nlag) * total_pixels + idx] - mean_image[idx];
            sum_corr += centered1 * centered2;
        }
        image_out[idx] = sum_corr / (frames - nlag);
    }
}

extern "C" {
    void temporal(const float* d_image_stack, float* d_image_out, float* d_mean_image, int type, int frames, int rows, int cols) {
        int total_pixels = rows * cols;
        int nlag = 1;

        // Define block and grid size
        dim3 blockSize(THREADS_PER_BLOCK);
        dim3 gridSize((total_pixels + blockSize.x - 1) / blockSize.x);

        if (type == 0) {
            average_kernel<<<gridSize, blockSize>>>(d_image_stack, d_image_out, frames, total_pixels);
        } else if (type == 1) {
            average_kernel<<<gridSize, blockSize>>>(d_image_stack, d_mean_image, frames, total_pixels);
            hipDeviceSynchronize();  // Ensure mean is computed before using it
            variance_kernel<<<gridSize, blockSize>>>(d_image_stack, d_image_out, d_mean_image, frames, total_pixels);
        } else {
            average_kernel<<<gridSize, blockSize>>>(d_image_stack, d_mean_image, frames, total_pixels);
            hipDeviceSynchronize();
            temporal_auto_correlation_kernel<<<gridSize, blockSize>>>(
                d_image_stack, d_image_out, d_mean_image, frames, total_pixels, nlag);
        }
    }
}
